#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil_inline.h>

template <int BLOCK_SIZE> __global__ void
matrixMulCUDA(float *C, float *A, float *B, int wA, int wB)
{
	    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {
        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

extern "C" int matrix_mul_cuda( float* h_C, float* h_A, float* h_B, int ra, int ca, int cb )
{
	int status = 0; // success
	int rb = ca;

	int block_size = 16;

	unsigned int mem_size_A = sizeof(float) *ra*ca;
	unsigned int mem_size_B = sizeof(float) *rb*cb;
	unsigned int mem_sie_C = sizeof(float) *ra*cb;
	
	float *d_A, *d_B, *d_C;
	
	hipError_t error;
	hipMalloc((void **) &d_A, mem_size_A);
	hipMalloc((void**) &d_B, mem_size_B);
	hipMalloc((void**) &d_C, mem_sie_C);

	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

	//ececute parameters
    dim3 threads(block_size, block_size);
    dim3 grid(cb / threads.x, ra / threads.y);

	matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, ca, cb);
	hipDeviceSynchronize();

	hipMemcpy(h_C, d_C, mem_sie_C, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

	return status;
}
