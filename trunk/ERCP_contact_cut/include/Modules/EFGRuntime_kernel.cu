#include "hip/hip_runtime.h"
#define GRAVITY 98
#define DIM 3
#define SDIM 6

#include <hip/hip_runtime.h>
#include <cutil_inline.h>

int  d_NbNode;
int	 d_NbNodeAdded;
int	 d_NbNodeInside;

float* d_NodePos0;
float* d_NodePos;
float* d_NodeVel;
float* d_NodeDis;
float* d_NodeForce;
float* d_NodeStrain;
float* d_NodeStress;
float* d_NodeVolume;
float* d_NodeMass;

int* d_NodeIdx;
int* d_NeighborNodeIdx;
int* d_NbNeighborNode;
int  d_NbNeighborMax;

float* d_ShapeFuncDerivAtNode;
float* d_ShapeFuncDerivXAtNode;
float* d_ShapeFuncDerivYAtNode;
float* d_ShapeFuncDerivZAtNode;
float* d_ShapeFuncDerivAtNodeInv;
float* d_ShapeFuncDerivXAtNodeInv;
float* d_ShapeFuncDerivYAtNodeInv;
float* d_ShapeFuncDerivZAtNodeInv;

// Stress point
int	 d_NbStressPoint;
float* d_StressVolume;
float* d_StressPos0;
float* d_StressPos;

int* d_NeighborStressPointIdx;
int* d_NbNeighborStressPoint;
int* d_StressNeighborNodeIdx;
int* d_NbStressNeighborNode;

float* d_ShapeFuncDerivAtStressPoint;
float* d_ShapeFuncDerivAtStressPointInv;

float* d_StressPointStrain;
float* d_StressPointStress;

float* d_MaterialStiffness;
int* d_FixedConstraint;

__global__ 
void updateValueFloat_k(float* previousValue, float* updatedValue, int* updatedIdx, int nbUpdated)
{
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	int _idx=updatedIdx[idx];
	if(idx<nbUpdated)
		previousValue[_idx]=updatedValue[idx];
}

__global__ 
void updateValueInt_k(int* previousValue, int* updatedValue, int* updatedIdx, int nbUpdated)
{
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	int _idx=updatedIdx[idx];
	if(idx<nbUpdated)
		previousValue[_idx]=updatedValue[idx];
}

__global__ 
void computeStrain_k(int* neighborNodeIdx, int* nbNeighborNode, int nbNeighborMax, float* shapeFuncDerivAtNodeInv, float* strain, float* nodeDis, int size)
{
	int nodeIdx = blockIdx.x;
	int tx = threadIdx.x;

	__shared__ float _strain[6];
	_strain[tx]=0;
		
	int idx1=tx%3;
	int idx2=tx-3;
	int idx3=(tx-2)%3;
	
	if(tx<3)
	{
		for(int i=0;i<nbNeighborNode[nodeIdx];i++)
		{
			float dis;
			int idx=nbNeighborMax*nodeIdx+i;
			dis=nodeDis[neighborNodeIdx[idx]*3+tx]-nodeDis[nodeIdx*3+tx];
			_strain[tx]+=shapeFuncDerivAtNodeInv[idx+size*tx]*dis;
		}
	}
	else
	{
		for(int i=0;i<nbNeighborNode[nodeIdx];i++)
		{
			float dis[2];
			int idx=nbNeighborMax*nodeIdx+i;
			dis[0]=nodeDis[neighborNodeIdx[idx]*3+idx2]-nodeDis[nodeIdx*3+idx2];
			dis[1]=nodeDis[neighborNodeIdx[idx]*3+idx3]-nodeDis[nodeIdx*3+idx3];
			_strain[tx]+=(shapeFuncDerivAtNodeInv[idx+size*idx3]*dis[0]+shapeFuncDerivAtNodeInv[idx+size*idx2]*dis[1]);
		}
	}
	strain[nodeIdx*6+tx]=_strain[tx];
}

__global__ 
void computeStress_k(float* nodeStrain, float* nodeStress, float* materialStiffness)
{
	int nodeIdx = blockIdx.x;
	int idx=threadIdx.x;
	
	__shared__ float stiffness[6][6];
	for(int i=0;i<6;i++)
	{
		stiffness[i][idx]=materialStiffness[6*i+idx];
	}

	__shared__ float strain[6];
	strain[idx]=nodeStrain[nodeIdx*6+idx];

	// Synchronize to make sure the matrices are loaded
    __syncthreads();

	nodeStress[nodeIdx*6+idx]=0;
	for(int i=0;i<6;i++)
	{
		nodeStress[nodeIdx*6+idx]+=stiffness[idx][i]*strain[idx];
	}
}

__global__ 
void computeForce_k(int* neighborNodeIdx, int* nbNeighborNode, int nbNeighborMax, float* shapeFuncDrv, float* stress, float* volume, float* vel, float* mass, float damping, float* force, int size)
{
	int nodeIdx = blockIdx.x;
	int tx = threadIdx.x;

	int idx1=size*tx;
	int idx2=size*((tx+1)%3);
	int idx3=size*((tx+2)%3);
	int idx4=(tx+2)%3+3;
	float _force=0;
	
	for(int i=0;i<nbNeighborNode[nodeIdx];i++)
	{
		int idx=nbNeighborMax*nodeIdx+i;
		_force-=(shapeFuncDrv[idx+idx1]*stress[neighborNodeIdx[idx]*6+tx]+shapeFuncDrv[idx+idx2]*stress[neighborNodeIdx[idx]*6+tx+3]+shapeFuncDrv[idx+idx3]*stress[neighborNodeIdx[idx]*6+idx4])*volume[nodeIdx];
	}

	//Add gravity
	if(tx==1)
		_force-=GRAVITY*mass[nodeIdx];

	//Add damping force
	_force-=vel[nodeIdx*3+tx]*damping;

	force[nodeIdx*3+tx]=_force;
}

__global__ 
void computeForce_k(int* neighborNodeIdx, int* nbNeighborNode, int nbNeighborMax, float* shapeFuncDrv, float* stress, float* volume, float* pos, float* vel, float* mass, float damping, float* force, int size)
{
	int nodeIdx = blockIdx.x;
	int tx = threadIdx.x;

	int idx1=size*tx;
	int idx2=size*((tx+1)%3);
	int idx3=size*((tx+2)%3);
	int idx4=(tx+2)%3+3;
	float _force=0;
	
	for(int i=0;i<nbNeighborNode[nodeIdx];i++)
	{
		int idx=nbNeighborMax*nodeIdx+i;
		_force-=(shapeFuncDrv[idx+idx1]*stress[neighborNodeIdx[idx]*6+tx]+shapeFuncDrv[idx+idx2]*stress[neighborNodeIdx[idx]*6+tx+3]+shapeFuncDrv[idx+idx3]*stress[neighborNodeIdx[idx]*6+idx4])*volume[nodeIdx];
	}

	//Add gravity
	if(tx==1)
		_force-=GRAVITY*mass[nodeIdx];

	//Add damping force
	_force-=vel[nodeIdx*3+tx]*damping;

	force[nodeIdx*3+tx]=_force;
}

__global__ 
void computeForceStress_k(int* neighborNodeIdx, int* nbNeighborNode, int nbNeighborMax, float* shapeFuncDrv, float* stress, float* volume, float* force, int size)
{
	int nodeIdx = blockIdx.x;
	int tx = threadIdx.x;

	int idx1=size*tx;
	int idx2=size*((tx+1)%3);
	int idx3=size*((tx+2)%3);
	int idx4=(tx+2)%3+3;
	float _force=0;
	
	for(int i=0;i<nbNeighborNode[nodeIdx];i++)
	{
		int idx=nbNeighborMax*nodeIdx+i;
		_force-=(shapeFuncDrv[idx+idx1]*stress[neighborNodeIdx[idx]*6+tx]+shapeFuncDrv[idx+idx2]*stress[neighborNodeIdx[idx]*6+tx+3]+shapeFuncDrv[idx+idx3]*stress[neighborNodeIdx[idx]*6+idx4])*volume[nodeIdx];
	}
	force[nodeIdx*3+tx]+=_force;
}

__global__ 
void explicitIntegration_k(float* dis, float* pos, float* pos0, float* vel, float* force, float* mass, int* constraint, int nbConstraint, float dt)
{
	int nodeIdx = blockIdx.x;
	int tx = threadIdx.x;

	float dv=force[nodeIdx*3+tx]/mass[nodeIdx]*dt;
	vel[nodeIdx*3+tx]+=dv;
	dis[nodeIdx*3+tx]+=(vel[nodeIdx*3+tx]*dt);
	pos[nodeIdx*3+tx]=pos0[nodeIdx*3+tx]+dis[nodeIdx*3+tx];
}

__global__ 
void explicitIntegrationConst_k(float* dis, float* pos, float* pos0, float* force, float* mass, float dt)
{
	int nodeIdx = blockIdx.x;
	int tx = threadIdx.x;

	float dv=force[nodeIdx*3+tx]/mass[nodeIdx]*dt;
	dis[nodeIdx*3+tx]+=(dv*dt);
	pos[nodeIdx*3+tx]=pos0[nodeIdx*3+tx]+dis[nodeIdx*3+tx];
}

__global__ 
void fixedConstraint_k(float* dis, float* pos, float* pos0, int* constraint)
{
	int idx=blockIdx.x;
	int tx=threadIdx.x;

	pos[constraint[idx]*3+tx]=pos0[constraint[idx]*3+tx];
	dis[constraint[idx]*3+tx]=0;
}

extern "C" void d_initGPU(int nbNode, int nbNodeAdded, float* nodeVolume, float* nodePos0, float* nodePos, float* nodeVel, int nbNodeInside)
{
	// Initialize
	int devID;
	hipDeviceProp_t props;
	d_NbNode=nbNode;
	d_NbNodeAdded=nbNodeAdded;
	d_NbNodeInside=nbNodeInside;
	d_NbStressPoint=-1;

	// get number of SMs on this GPU
	cutilSafeCall(hipGetDevice(&devID));
	cutilSafeCall(hipGetDeviceProperties(&props, devID));

	// Memory allocation
	int size=(nbNode+nbNodeAdded)*sizeof(float);
	cutilSafeCall(hipMalloc((void**) &d_NodeVolume, size));
	cutilSafeCall(hipMemcpy(d_NodeVolume, nodeVolume, size, hipMemcpyHostToDevice) );

	size=nbNode*DIM*sizeof(float);
	cutilSafeCall(hipMalloc((void**) &d_NodePos0, size));
	cutilSafeCall(hipMalloc((void**) &d_NodePos, size));
	cutilSafeCall(hipMalloc((void**) &d_NodeVel, size));
	cutilSafeCall(hipMemcpy(d_NodePos0, nodePos0, size, hipMemcpyHostToDevice) );
	cutilSafeCall(hipMemcpy(d_NodePos, nodePos, size, hipMemcpyHostToDevice) );
	cutilSafeCall(hipMemcpy(d_NodeVel, nodeVel, size, hipMemcpyHostToDevice) );

	cutilSafeCall(hipMalloc((void**) &d_NodeDis, size));
	cutilSafeCall(hipMalloc((void**) &d_NodeForce, size));
	// zero default value for nodeDis, nodeForce, nodeStrain and nodeStress
	{
		float* temp0 = (float*)malloc(size);
		int i=0;
		for(i=0; i<nbNode*DIM; i++)
		{
			temp0[i] = 0;
		}
		cutilSafeCall(hipMemcpy(d_NodeDis, temp0, size, hipMemcpyHostToDevice) );
		cutilSafeCall(hipMemcpy(d_NodeForce, temp0, size, hipMemcpyHostToDevice) );
		free(temp0);
	}

	size=nbNode*SDIM*sizeof(float);
	cutilSafeCall(hipMalloc((void**) &d_NodeStrain, size));
	cutilSafeCall(hipMalloc((void**) &d_NodeStress, size));
		// zero default value for nodeDis, nodeForce, nodeStrain and nodeStress
	{
		float* temp0 = (float*)malloc(size);
		int i=0;
		for(i=0; i<nbNode*SDIM; i++)
		{
			temp0[i] = 0;
		}
		cutilSafeCall(hipMemcpy(d_NodeStrain, temp0, size, hipMemcpyHostToDevice) );
		cutilSafeCall(hipMemcpy(d_NodeStress, temp0, size, hipMemcpyHostToDevice) );
		free(temp0);
	}
}

extern "C" void d_initStressPoint(int nbStress, float* stressVolume, float* stressPos0, float* stressPos)
{
	d_NbStressPoint=nbStress;
	
	// Memory allocation
	int size=(d_NbStressPoint+d_NbNodeAdded)*sizeof(float);
	cutilSafeCall(hipMalloc((void**) &d_StressVolume, size));
	cutilSafeCall(hipMemcpy(d_StressVolume, stressVolume, size, hipMemcpyHostToDevice));
	
	size=nbStress*DIM*sizeof(float);
	cutilSafeCall(hipMalloc((void**) &d_StressPos0, size));
	cutilSafeCall(hipMalloc((void**) &d_StressPos, size));
	cutilSafeCall(hipMemcpy(d_StressPos0, stressPos0, size, hipMemcpyHostToDevice) );
	cutilSafeCall(hipMemcpy(d_StressPos, stressPos, size, hipMemcpyHostToDevice) );
	
	size=nbStress*SDIM*sizeof(float);
	cutilSafeCall(hipMalloc((void**) &d_StressPointStrain, size));
	cutilSafeCall(hipMalloc((void**) &d_StressPointStress, size));
}

extern "C" void d_initNeighborInformation(int* neighborNodeIdx, int* nbNeighborNode, int nbNeighborMax)
{
	int size=(d_NbNode+d_NbNodeAdded)*sizeof(int);
	cutilSafeCall(hipMalloc((void**) &d_NodeIdx, size));

	size=nbNeighborMax*(d_NbNode+d_NbNodeAdded)*sizeof(int);
	cutilSafeCall(hipMalloc((void**) &d_NeighborNodeIdx, size));
	cutilSafeCall(hipMemcpy(d_NeighborNodeIdx, neighborNodeIdx, size, hipMemcpyHostToDevice));

	size=(d_NbNode+d_NbNodeAdded)*sizeof(int);
	cutilSafeCall(hipMalloc((void**) &d_NbNeighborNode, size));

	cutilSafeCall(hipMemcpy(d_NbNeighborNode, nbNeighborNode, size, hipMemcpyHostToDevice));
	d_NbNeighborMax=nbNeighborMax;
}

extern "C" void d_initNeighborInformationStress(int* neighborStressPointIdx, int* nbNeighborStressPoint, int* stressNeighborNodeIdx, int* nbStressNeighborNode)
{
	int size=d_NbNeighborMax*(d_NbNode+d_NbNodeAdded)*sizeof(int);
	cutilSafeCall(hipMalloc((void**) &d_NeighborStressPointIdx, size));
	cutilSafeCall(hipMemcpy(d_NeighborStressPointIdx, neighborStressPointIdx, size, hipMemcpyHostToDevice));
	
	size=(d_NbNode+d_NbNodeAdded)*sizeof(int);
	cutilSafeCall(hipMalloc((void**) &d_NbNeighborStressPoint, size));
	cutilSafeCall(hipMemcpy(d_NbNeighborStressPoint, nbNeighborStressPoint, size, hipMemcpyHostToDevice));
	
	size=d_NbNeighborMax*(d_NbStressPoint+d_NbNodeAdded)*sizeof(int);
	cutilSafeCall(hipMalloc((void**) &d_StressNeighborNodeIdx, size));
	cutilSafeCall(hipMemcpy(d_StressNeighborNodeIdx, stressNeighborNodeIdx, size, hipMemcpyHostToDevice));
	
	size=(d_NbStressPoint+d_NbNodeAdded)*sizeof(int);
	cutilSafeCall(hipMalloc((void**) &d_NbStressNeighborNode, size));
	cutilSafeCall(hipMemcpy(d_NbStressNeighborNode, nbStressNeighborNode, size, hipMemcpyHostToDevice));
}

extern "C" void d_initShapeFuncValue(float* shapeFuncDerivAtNode, float* shapeFuncDerivAtNodeInv)
{
	int size=(d_NbNode+d_NbNodeAdded)*d_NbNeighborMax*sizeof(float)*3;
	cutilSafeCall(hipMalloc((void**) &d_ShapeFuncDerivAtNode, size));
	cutilSafeCall(hipMalloc((void**) &d_ShapeFuncDerivAtNodeInv, size));

	cutilSafeCall(hipMemcpy(d_ShapeFuncDerivAtNode, shapeFuncDerivAtNode, size, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_ShapeFuncDerivAtNodeInv, shapeFuncDerivAtNodeInv, size, hipMemcpyHostToDevice));
}

extern "C" void d_initShapeFuncValueStressPoint(float* shapeFuncDerivAtStressPoint, float* shapeFuncDerivAtStressPointInv)
{
	int size=(d_NbNode+d_NbNodeAdded)*d_NbNeighborMax*sizeof(float)*3;
	cutilSafeCall(hipMalloc((void**) &d_ShapeFuncDerivAtStressPoint, size));
	cutilSafeCall(hipMemcpy(d_ShapeFuncDerivAtStressPoint, shapeFuncDerivAtStressPoint, size, hipMemcpyHostToDevice));
	
	size=(d_NbStressPoint+d_NbNodeAdded)*d_NbNeighborMax*sizeof(float)*3;
	cutilSafeCall(hipMalloc((void**) &d_ShapeFuncDerivAtStressPointInv, size));
	cutilSafeCall(hipMemcpy(d_ShapeFuncDerivAtStressPointInv, shapeFuncDerivAtStressPointInv, size, hipMemcpyHostToDevice));
}


extern "C" void d_initMaterialStiffness(float* materialStiffness, float* nodeMass)
{
	hipError_t error;
	int size=SDIM*SDIM*sizeof(float);
	cutilSafeCall(hipMalloc((void**) &d_MaterialStiffness, size));
	cutilSafeCall(hipMemcpy(d_MaterialStiffness, materialStiffness, size, hipMemcpyHostToDevice));

	size=(d_NbNode+d_NbNodeAdded)*sizeof(float);
	cutilSafeCall(hipMalloc((void**) &d_NodeMass, size));
	cutilSafeCall(hipMemcpy(d_NodeMass, nodeMass, size, hipMemcpyHostToDevice));
}

extern "C" void d_explicitIntegration(float dt, int nbFixedConstraint)
{
	dim3 threadsPerBlock(DIM);
	dim3 blocksPerGrid(d_NbNode);

	explicitIntegration_k<<< blocksPerGrid, threadsPerBlock >>> (d_NodeDis, d_NodePos, d_NodePos0, d_NodeVel, d_NodeForce, d_NodeMass, d_FixedConstraint, nbFixedConstraint, dt);

	if(nbFixedConstraint > 0)
	{
		blocksPerGrid.x = nbFixedConstraint;
		fixedConstraint_k<<< blocksPerGrid, threadsPerBlock >>> (d_NodeDis, d_NodePos, d_NodePos0, d_FixedConstraint);
	}
}

extern "C" void d_explicitIntegrationConst(float dt)
{
	dim3 threadsPerBlock(DIM);
	dim3 blocksPerGrid(d_NbNode);
	
	explicitIntegrationConst_k<<< blocksPerGrid, threadsPerBlock >>> (d_NodeDis, d_NodePos, d_NodePos0, d_NodeForce, d_NodeMass, dt);
}

extern "C" void d_computeStrain()
{
	dim3 threadsPerBlock(SDIM);
	dim3 blocksPerGrid(d_NbNode);
	
	int size=(d_NbNode+d_NbNodeAdded)*d_NbNeighborMax;
	computeStrain_k<<< blocksPerGrid, threadsPerBlock >>>(d_NeighborNodeIdx, d_NbNeighborNode, d_NbNeighborMax, d_ShapeFuncDerivAtNodeInv, d_NodeStrain, d_NodeDis, size);
	
	if(d_NbStressPoint>0)
	{
		size=(d_NbStressPoint+d_NbNodeAdded)*d_NbNeighborMax;	
		blocksPerGrid.x=d_NbStressPoint;
		computeStrain_k<<< blocksPerGrid, threadsPerBlock >>>(d_StressNeighborNodeIdx, d_NbStressNeighborNode, d_NbNeighborMax, d_ShapeFuncDerivAtStressPointInv, d_StressPointStrain, d_NodeDis, size);
	}	
}

extern "C" void d_computeStress()
{
	dim3 threadsPerBlock(SDIM);
	dim3 blocksPerGrid(d_NbNode);

	computeStress_k<<< blocksPerGrid, threadsPerBlock >>>(d_NodeStrain, d_NodeStress, d_MaterialStiffness);
	
	if(d_NbStressPoint>0)
	{
		blocksPerGrid.x=d_NbStressPoint;
		computeStress_k<<< blocksPerGrid, threadsPerBlock >>>(d_StressPointStrain, d_StressPointStress, d_MaterialStiffness);
	}
}

extern "C" void d_computeForce(float damping)
{
	dim3 threadsPerBlock(DIM);
	dim3 blocksPerGrid(d_NbNode);

	int size=(d_NbNode+d_NbNodeAdded)*d_NbNeighborMax;
	computeForce_k<<< blocksPerGrid, threadsPerBlock >>>(d_NeighborNodeIdx, d_NbNeighborNode, d_NbNeighborMax, d_ShapeFuncDerivAtNode, d_NodeStress, d_NodeVolume, d_NodePos0, d_NodeVel, d_NodeMass, damping, d_NodeForce, size);
	
	if(d_NbStressPoint>0)
	{
		computeForceStress_k<<< blocksPerGrid, threadsPerBlock >>>(d_NeighborStressPointIdx, d_NbNeighborStressPoint, d_NbNeighborMax, d_ShapeFuncDerivAtStressPoint, d_StressPointStress, d_StressVolume, d_NodeForce, size);
	}
}

extern "C" void d_initFixedConstraint(int nbFixedConstraint, int* fixedIdx)
{
	int size=nbFixedConstraint*sizeof(int);
	cutilSafeCall(hipMalloc((void**) &d_FixedConstraint, size));
	cutilSafeCall(hipMemcpy(d_FixedConstraint, fixedIdx, size, hipMemcpyHostToDevice));
}

extern "C" void d_getPosition(float* nodePos)
{
	int size=d_NbNode*DIM*sizeof(float);
	cutilSafeCall(hipMemcpy(nodePos, d_NodePos, size, hipMemcpyDeviceToHost));
}

extern "C" void d_getDisplacement(float* nodeDis)
{
	int size=d_NbNode*DIM*sizeof(float);
	cutilSafeCall(hipMemcpy(nodeDis, d_NodeDis, size, hipMemcpyDeviceToHost));
}

extern "C" void d_getMass(float* nodeMass)
{
	int size=d_NbNode*sizeof(float);
	cutilSafeCall(hipMemcpy(nodeMass, d_NodeMass, size, hipMemcpyDeviceToHost));
}

extern "C" void d_getStressPointStrain(float* stressPointStrain)
{
	int size=d_NbStressPoint*sizeof(float)*SDIM;
	cutilSafeCall(hipMemcpy(stressPointStrain, d_StressPointStrain, size, hipMemcpyDeviceToHost));
}
extern "C" void d_getStressPointStress(float* stressPointStress)
{
	int size=d_NbStressPoint*sizeof(float)*SDIM;
	cutilSafeCall(hipMemcpy(stressPointStress, d_StressPointStress, size, hipMemcpyDeviceToHost));
}
extern "C" void d_getNodeStrain(float* nodeStrain)
{
	int size=d_NbNode*sizeof(float)*SDIM;
	cutilSafeCall(hipMemcpy(nodeStrain, d_NodeStrain, size, hipMemcpyDeviceToHost));
}
extern "C" void d_getNodeStress(float* nodeStress)
{
	int size=d_NbStressPoint*sizeof(float)*SDIM;
	cutilSafeCall(hipMemcpy(nodeStress, d_NodeStress, size, hipMemcpyDeviceToHost));
}

extern "C" void d_setForce(float* nodeForce)
{
	int size=d_NbNode*sizeof(float)*DIM;
	cutilSafeCall(hipMemcpy(d_NodeForce, nodeForce, size, hipMemcpyHostToDevice));
}

extern "C" void d_updateNbNeighborNodeFull(int* nbNeighborNode, int size)
{
	hipMemcpy(d_NbNeighborNode, nbNeighborNode, size, hipMemcpyHostToDevice);
}
extern "C" void d_updateNbNeighborNodePartial(int* updatedValue, int* updatedIdx, int nbUpdated)
{
	int size=nbUpdated*sizeof(int);
	int* d_UpdatedValue;
	int* d_UpdatedIdx;
	hipMalloc((void**) &d_UpdatedValue, size);
	hipMalloc((void**) &d_UpdatedIdx, size);
	hipMemcpy(d_UpdatedValue, updatedValue, size, hipMemcpyHostToDevice);
	hipMemcpy(d_UpdatedIdx, updatedIdx, size, hipMemcpyHostToDevice);

	int nbBlock=nbUpdated/256+1;
	dim3 threadsPerBlock(256);
	dim3 blocksPerGrid(nbBlock);
	updateValueInt_k<<< blocksPerGrid, threadsPerBlock >>> (d_NbNeighborNode, updatedValue, updatedIdx, nbUpdated);
}
extern "C" void d_updateNeighborNodeIdxFull(int* neighborNodeIdx, int size)
{
	hipMemcpy(d_NeighborNodeIdx, neighborNodeIdx, size, hipMemcpyHostToDevice);
}
extern "C" void d_updateNeighborNodeIdxPartial(int* updatedValue, int* updatedIdx, int nbUpdated)
{
	int size=nbUpdated*sizeof(int);
	int* d_UpdatedValue;
	int* d_UpdatedIdx;
	hipMalloc((void**) &d_UpdatedValue, size);
	hipMalloc((void**) &d_UpdatedIdx, size);
	hipMemcpy(d_UpdatedValue, updatedValue, size, hipMemcpyHostToDevice);
	hipMemcpy(d_UpdatedIdx, updatedIdx, size, hipMemcpyHostToDevice);

	int nbBlock=nbUpdated/256+1;
	dim3 threadsPerBlock(256);
	dim3 blocksPerGrid(nbBlock);
	updateValueInt_k<<< blocksPerGrid, threadsPerBlock >>> (d_NeighborNodeIdx, d_UpdatedValue, d_UpdatedIdx, nbUpdated);
}

extern "C" void d_updateShapeFuncDrvFull(float* value, int size)
{	
	hipMemcpy(d_ShapeFuncDerivAtNode, value, size, hipMemcpyHostToDevice);
}

extern "C" void d_updateShapeFuncDrvInvFull(float* value, int size)
{	
	hipMemcpy(d_ShapeFuncDerivAtNodeInv, value, size, hipMemcpyHostToDevice);
}

extern "C" void d_updateShapeFuncDrvPartial(float* updatedValue, int* updatedIdx, int nbUpdated)
{
	int size=nbUpdated*sizeof(float);
	float* d_UpdatedValue;
	hipMalloc((void**) &d_UpdatedValue, size);
	hipMemcpy(d_UpdatedValue, updatedValue, size, hipMemcpyHostToDevice);

	size=nbUpdated*sizeof(int);
	int* d_UpdatedIdx;
	hipMalloc((void**) &d_UpdatedIdx, size);
	hipMemcpy(d_UpdatedIdx, updatedIdx, size, hipMemcpyHostToDevice);

	int nbBlock=nbUpdated/256+1;
	dim3 threadsPerBlock(256);
	dim3 blocksPerGrid(nbBlock);
	updateValueFloat_k<<< blocksPerGrid, threadsPerBlock >>> (d_ShapeFuncDerivAtNode, d_UpdatedValue, d_UpdatedIdx, nbUpdated);
}

extern "C" void d_updateShapeFuncDrvInvPartial(float* updatedValue, int* updatedIdx, int nbUpdated)
{
	int size=nbUpdated*sizeof(float);
	float* d_UpdatedValue;
	hipMalloc((void**) &d_UpdatedValue, size);
	hipMemcpy(d_UpdatedValue, updatedValue, size, hipMemcpyHostToDevice);

	size=nbUpdated*sizeof(int);
	int* d_UpdatedIdx;
	hipMalloc((void**) &d_UpdatedIdx, size);
	hipMemcpy(d_UpdatedIdx, updatedIdx, size, hipMemcpyHostToDevice);

	int nbBlock=nbUpdated/256+1;
	dim3 threadsPerBlock(256);
	dim3 blocksPerGrid(nbBlock);
	updateValueFloat_k<<< blocksPerGrid, threadsPerBlock >>> (d_ShapeFuncDerivAtNodeInv, d_UpdatedValue, d_UpdatedIdx, nbUpdated);
}